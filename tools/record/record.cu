#include "hip/hip_runtime.h"
/* 
 *
 * A NVBit tool, which will log the input and output of kernel function
 * input is defined as memory copied from host to device and arguments of kernel
 * output is defined as memory copied from device to host
 *
 * And it will also record memory access of each thread for selected instructions.
 * code that related to the use of channel credit to nvbit sample tool "mem_trace"
 *
 * Yineng Yan (yinengy@umich.edu), 2020
 */
#include <stdio.h>
#include <string>
#include <vector>
#include <iostream>
#include <fstream>
#include <cerrno>
#include <algorithm>
#include <sys/stat.h>
#include <assert.h>
#include <unordered_set>
#include <stdint.h>
#include <map>

/* serialize vector */
#include <cereal/archives/binary.hpp>
#include <cereal/types/vector.hpp>

/* header for every nvbit tool */
#include "nvbit_tool.h"

/* interface of nvbit */
#include "nvbit.h"

/* nvbit utility functions */
#include "utils/utils.h"

/* for channel */
#include "utils/channel.hpp"

/* definition of the mem_access_t structure */
#include "common.h"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

/* counter for hipMemcpy and kernel launch */
int cudaMemcpy_input_count = 0;
int cudaMemcpy_output_count = 0;
int funcParams_count = 0;

/* 3D vector of [block_id, thread_id, num_access] to value of memory accesses */
std::vector<std::vector<std::vector<uint32_t>>> mem_val;

/* map of func_id to set of inst_id */
std::map<int, std::unordered_set<int>> data_race_log;

/* counter for func_id */
int func_counter = 0;


/* will save memory to files 
 * if is_input is 0, it means the mem is copied from device to host
 * otherwise, the mem is copied from host to device
 */
void dump_mem(const void *src, size_t ByteCount, int is_input) {
    
    char filename[25]; // large enough for a counter

    if (is_input) {
        cudaMemcpy_input_count++;
        sprintf(filename, "kernel_log/imem%d.bin", cudaMemcpy_input_count);
    } else {
        cudaMemcpy_output_count++;
        sprintf(filename, "kernel_log/omem%d.bin", cudaMemcpy_output_count);
    }
    
    std::ofstream file(filename, std::ios::out | std::ios::binary);

    if (!file.is_open()) {
        std::cerr << strerror(errno) << "failed to open file.\n";
        exit(1);
    }

    file.write((char *) src, ByteCount);

    file.close();
}

void get_data_race_log() {
    std::ifstream file;
    file.open("datarace.txt");

    if (file.fail()) {
        std::cerr << strerror(errno) << "failed to open file.\n";
        exit(1);
    }

    int func_id, inst_id;
    char comma;
    std::string line;
    while (std::getline(file, line)) {
        /* read type */
        std::istringstream iss(line);
        iss >> func_id;
        iss >> comma;
        iss >> inst_id;

        if (data_race_log.find(func_id) == data_race_log.end()) {
            data_race_log[func_id] = std::unordered_set<int>();
        } 
        
        data_race_log[func_id].insert(inst_id);
    }
}

/* save arguments to files
 * if it is not a pointer, its value with be saved
 * TODO: user defined type is not support
 */
void save_nonpointer_arguments(void **kernelParams,std::string func_sig) {
    /* get range of parameters */
    size_t begin = func_sig.find_first_of("(") + 1; // left parenthesis
    size_t end = func_sig.find_last_of(")"); // right parenthesis

    /* get string of parameters from string of signature */
    std::string func_params_str = func_sig.substr(begin, end - begin);

    /* remove whitespace */
    func_params_str.erase(remove_if(func_params_str.begin(), func_params_str.end(), isspace), func_params_str.end());

    /* split by comma */
    std::vector<std::string> func_params;
    std::string delim = ",";
    begin = 0;
    end = func_params_str.find(delim);
    while (end != std::string::npos) {
        func_params.push_back(func_params_str.substr(begin, end - begin));
        begin = end + delim.length();
        end = func_params_str.find(delim, begin);
    }
    func_params.push_back(func_params_str.substr(begin, end));

    /* check types of parameters
     * if it is not a pointer (pointer will be processed separately)
     * save its index, type and value to file
     * if it is a pointer, its value will be marked as "POINTER"
     */ 
    funcParams_count++;
    char filename[25];
    sprintf(filename, "kernel_log/param%d.txt", funcParams_count);
    std::ofstream file;
    file.open(filename);

    if (file.fail()) {
        std::cerr << strerror(errno) << "failed to open file.\n";
        exit(1);
    }

    std::string type;
    for (size_t i = 0; i < func_params.size(); i++) {
        type = func_params[i];
        if (type.empty()) { 
            // the function has no parameter
            break;
        } 

        file << i << "," << type << ",";

        /* cast kernelParams based on parameter type
         * it will gives the argument of the kernel function
         * refer to https://en.wikipedia.org/wiki/C_data_types
         */
        if (type.find('*') != std::string::npos) {
            // the parameter is a pointer
            file << "POINTER"; 
        } else if (type == "char") {
            file << ((char **) kernelParams)[i][0]; 
        } else if (type == "signedchar") {
            file << ((signed char **) kernelParams)[i][0]; 
        } else if (type == "unsignedchar") {
            file << ((unsigned char **) kernelParams)[i][0]; 
        } else if (type == "short" ||
                   type == "shortint" ||
                   type == "signedshort" ||
                   type == "signedshortint") {
            // signed short
            file << ((short **) kernelParams)[i][0]; 
        } else if (type == "unsignedshort" ||
                   type == "unsigned short int") {
            // unsigned short
            file << ((unsigned short **) kernelParams)[i][0]; 
        } else if (type == "int" ||
                   type == "signed" ||
                   type == "signedint") {
            // signed int
            file << ((int **) kernelParams)[i][0]; 
        } else if (type == "unsigned" ||
                   type == "unsignedint") {
            // unsigned int
            file << ((unsigned **) kernelParams)[i][0]; 
        } else if (type == "long" ||
                   type == "longint" ||
                   type == "signedlong" ||
                   type == "signedlongint") {
            // signed long
            file << ((long **) kernelParams)[i][0]; 
        } else if (type == "unsignedlong" ||
                   type == "unsignedlongint") {
            // unsigned long
            file << ((unsigned long **) kernelParams)[i][0]; 
        } else if (type == "longlong" ||
                   type == "longlongint" ||
                   type == "signedlonglong" ||
                   type == "signedlonglongint") {
            // signed long long
            file << ((long long **) kernelParams)[i][0]; 
        } else if (type == "unsignedlonglong" ||
                   type == "unsignedlonglongint") {
            // unsigned long long
            file << ((unsigned long long **) kernelParams)[i][0]; 
        } else if (type == "float") {
            // float
            file << ((float **) kernelParams)[i][0]; 
        } else if (type == "double") {
            // double
            file << ((double **) kernelParams)[i][0]; 
        } else if (type == "longdouble") {
            // long double
            file << ((long double **) kernelParams)[i][0]; 
        } else {
            // TODO: implement more types
            file << "UNKNOWN";
        }

        file << "\n";
    }

    file.close();
}

__global__ void flush_channel() {
    /* push memory access with negative block id to communicate the kernel is
     * completed */
    mem_access_t ma;
    ma.block_id = -1;
    channel_dev.push(&ma, sizeof(mem_access_t));

    /* flush channel */
    channel_dev.flush();
}

void *recv_thread_fun(void *) {
    char *recv_buffer = (char *)malloc(CHANNEL_SIZE);

    while (recv_thread_started) {
        uint32_t num_recv_bytes = 0;
        if (recv_thread_receiving &&
            (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
                0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {
                mem_access_t *ma =
                    (mem_access_t *)&recv_buffer[num_processed_bytes];

                /* when we get this cta_id_x it means the kernel has completed
                 */
                if (ma->block_id == -1) {
                    recv_thread_receiving = false;
                    break;
                }

                /* save memory value to vector */
                for (int i = 0; i < 32; i++) {   
                    if ((ma->active_mask >> i) & 1) { // if the thread is active
                        mem_val[ma->block_id][ma->l_thread_id + i].push_back(ma->vals[i]);
                    }
                }

                num_processed_bytes += sizeof(mem_access_t);
            }
        }
    }
    free(recv_buffer);
    return NULL;
}

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

        /* insert function name into the vector*/
        /* its index is func_id */
        int func_id = func_counter;
        func_counter++;

        int inst_id = 0;

        /* check if there is data race in this function */
        if (data_race_log.find(func_id) == data_race_log.end()) {
            continue; // no data race, go to next function
        }

        std::unordered_set<int> data_race_inst = data_race_log[func_id]; 

        /* iterate on all the static instructions in the function */
        for (auto instr : instrs) {
            /* check if there is data race in this instruction */
            if (data_race_inst.find(inst_id) == data_race_inst.end()) {
                inst_id++;
                continue; // no data race, go to next inst
            }

            std::string opcode = instr->getOpcodeShort();

            int reg_idx = 0;

            if (opcode == "LDG") {  // load
                /* it should has two operands */
                assert(instr->getNumOperands() == 2);

                /* first reg is the destination */
                const Instr::operand_t *dst_reg = instr->getOperand(0);
                assert(dst_reg->type == Instr::operandType::REG);
                reg_idx = dst_reg->u.reg.num;

                /* after this instruction so dst reg has the value */
                nvbit_insert_call(instr, "record_mem_val", IPOINT_AFTER);
            } else if (opcode == "STG") {  // store
                /* it should has two operands */
                assert(instr->getNumOperands() == 2);

                /* second reg is the source */
                const Instr::operand_t *src_reg = instr->getOperand(1);
                assert(src_reg->type == Instr::operandType::REG);
                reg_idx = src_reg->u.reg.num;

                /* before this instruction so src reg is not changed */
                nvbit_insert_call(instr, "record_mem_val", IPOINT_BEFORE);
            } else {
                // TODO: support more load and store instructions
                std::cerr << opcode << ": unhandled instruction.\n";
                instr->printDecoded();
                exit(1);
            }

            /* predicate value */
            nvbit_add_call_arg_pred_val(instr);
            /* add pointer to channel_dev*/
            nvbit_add_call_arg_const_val64(instr, (uint64_t)&channel_dev);
            /* add reg index */
            nvbit_add_call_arg_const_val32(instr, reg_idx, false);
            

            inst_id++;
        }
    }
}

/* This is triggered every time a hipMemcpy is called */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    if (skip_flag) return;

    /* if hipMemcpy from Device to Host
     * it is treated as output of the kernel
     * should be trigger at the end of cuMemcpy
     * so that dstHost has the memory ready
     */
    if ((cbid == API_CUDA_cuMemcpyDtoH_v2) && is_exit)  {
        /* get parameters */
        cuMemcpyDtoH_v2_params *p = (cuMemcpyDtoH_v2_params *)params;

        dump_mem(p->dstHost, p->ByteCount, 0);
    } else if ((cbid == API_CUDA_cuMemcpyHtoD_v2) && is_exit)  {
        cuMemcpyHtoD_v2_params *p = (cuMemcpyHtoD_v2_params *)params;

        dump_mem(p->srcHost, p->ByteCount, 1);
    } else if ((cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel)) {
        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

        if (!is_exit) {
            /* get kernel function signature */    
            std::string func_sig(nvbit_get_func_name(ctx, p->f));

            /* record kernel arguments */
            save_nonpointer_arguments(p->kernelParams, func_sig);

            /* instrument this kernel */
            instrument_function_if_needed(ctx, p->f);

            nvbit_enable_instrumented(ctx, p->f, true);

            /* reserve space */
            int num_block = p->gridDimX * p->gridDimY * p->gridDimZ;
            int num_thread = p->blockDimX * p->blockDimY * p->blockDimZ;
            mem_val.empty();
            mem_val.resize(num_block, std::vector<std::vector<uint32_t>>(num_thread));

            recv_thread_receiving = true;
        } else {
            /* make sure current kernel is completed */
            hipDeviceSynchronize();
            assert(hipGetLastError() == hipSuccess);

            /* make sure we prevent re-entry on the nvbit_callback when issuing
             * the flush_channel kernel */
            skip_flag = true;

            /* issue flush of channel so we are sure all the memory accesses
             * have been pushed */
            flush_channel<<<1, 1>>>();
            hipDeviceSynchronize();
            assert(hipGetLastError() == hipSuccess);

            /* unset the skip flag */
            skip_flag = false;

            /* wait here until the receiving thread has not finished with the
             * current kernel */
            while (recv_thread_receiving) {
                pthread_yield();
            }

            /* serilize vector */
            char filename[25];
            sprintf(filename, "kernel_log/vmem%d.bin", funcParams_count);
            std::ofstream file;
            file.open(filename);

            if (file.fail()) {
                std::cerr << strerror(errno) << "failed to open file.\n";
                exit(1);
            }

            {
                cereal::BinaryOutputArchive oarchive(file);

                oarchive(mem_val);
            }
        }
    }
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
    /* all log files will put into this directory */
    mkdir("kernel_log", 0777);

    /* initialize channel */
    recv_thread_started = true;
    channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
    pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);

    /* read data race log */
    get_data_race_log();
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    if (recv_thread_started) {
        recv_thread_started = false;
        pthread_join(recv_thread, NULL);
    }
}