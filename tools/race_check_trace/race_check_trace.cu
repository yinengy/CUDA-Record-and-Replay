#include "hip/hip_runtime.h"
/* 
 * An NVBit tool, which will detect conflict memory access in the kernel.
 * The raw output will be processed by a Pytyhon script
 *
 * The tool is based on thetool (mem_trace) in nvbit_release
 * the original code is modified and extended to support the use of detecting data races
 *
 * Yineng Yan (yinengy@umich.edu), 2020
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <string>
#include <map>
#include <unordered_set>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "common.h"

/* contains definition used for checking data race */
#include "race_checker.hpp"

/* output debug information of not */
#define DEBUG 0

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* synchronization operation counter, updated by the GPU threads */
int *syn_ops_counter = 0;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* race checker */
Checker race_checker;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

/* vector of func_name, index by func_id */
std::vector<std::string> id_to_func_name;

void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
}
/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
        const char*func_name = nvbit_get_func_name(ctx, f);
        if (verbose) {
            printf("Inspecting function %s at address 0x%lx\n",
                   func_name, nvbit_get_func_addr(f));
        }

        /* insert function name into the vector*/
        /* its index is func_id */
        int func_id = id_to_func_name.size();
        id_to_func_name.push_back(std::string(func_name));

        uint32_t inst_id = 0;

        /* tell python script the content of a function begins here*/
        printf("\n#func_begin#%s\n", func_name);

        /* iterate on all the static instructions in the function */
        for (auto instr : instrs) {
            // check syn op first
            const char *shortOpcode = instr->getOpcodeShort();

            // to see if it is a synchronization operation
            bool is_syn_op = (strcmp(shortOpcode, "RED") == 0)   || // Atomic Memory Reduction Operation
                             (strcmp(shortOpcode, "ATOM") == 0)  || // Atomic Operation on generic Memory
                             (strcmp(shortOpcode, "ATOMS") == 0) || // Atomic Operation on Shared Memory
                             (strcmp(shortOpcode, "BAR") == 0) ||   // Barrier (e.g. __syncthreads())
                             (strcmp(shortOpcode, "MEMBAR") == 0);  // Memory Barrier
            if (is_syn_op) {
                // instrument after the syn op so all thread are ready
                // then we can just update the counter by one
                nvbit_insert_call(instr, "instrument_syn", IPOINT_AFTER);
                nvbit_add_call_arg_const_val64(instr, (uint64_t)syn_ops_counter);
                inst_id++;
                continue; //skip the rest
            } 

            if (inst_id < instr_begin_interval || inst_id >= instr_end_interval ||
                ((instr->getMemOpType()!=Instr::memOpType::GLOBAL
                    && instr->getMemOpType()!=Instr::memOpType::SHARED && instr->getMemOpType()!=Instr::memOpType::GENERIC))) {
                inst_id++;
                continue;
            }
            if (verbose) {
                instr->printDecoded();
            }

            if (opcode_to_id_map.find(instr->getOpcode()) ==
                opcode_to_id_map.end()) {
                int opcode_id = opcode_to_id_map.size();
                opcode_to_id_map[instr->getOpcode()] = opcode_id;
                id_to_opcode_map[opcode_id] = std::string(instr->getOpcode());
            }

            int opcode_id = opcode_to_id_map[instr->getOpcode()];

            /* iterate on the operands */
            for (int i = 0; i < instr->getNumOperands(); i++) {
                /* get the operand "i" */
                const Instr::operand_t *op = instr->getOperand(i);
            
                if (op->type == Instr::operandType::MREF) {
                    /* insert call to the instrumentation function with its
                     * arguments */
                    nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
                    /* predicate value */
                    nvbit_add_call_arg_pred_val(instr);
                    /* func id */
                    nvbit_add_call_arg_const_val32(instr, func_id);    
                    /* inst id */
                    nvbit_add_call_arg_const_val32(instr, inst_id); 
                    /* memory reference 64 bit address */
                    nvbit_add_call_arg_mref_addr64(instr);
                    /* add pointer to channel_dev*/
                    nvbit_add_call_arg_const_val64(instr,
                                                   (uint64_t)&channel_dev);
                    // TODO: need to consider the case that memOpType is GENERIC
                    nvbit_add_call_arg_const_val32(instr, instr->getMemOpType()==Instr::memOpType::SHARED);
                    nvbit_add_call_arg_const_val32(instr, instr->isLoad());
                    nvbit_add_call_arg_const_val64(instr, (uint64_t)syn_ops_counter);
                }
            }
            inst_id++;
        }

        /* tell python script the content of a function ends here*/
        printf("\n#func_end#\n");
    }
}

__global__ void flush_channel() {
    /* push memory access with negative cta id to communicate the kernel is
     * completed */
    mem_access_t ma;
    ma.block_id = -1;
    channel_dev.push(&ma, sizeof(mem_access_t));

    /* flush channel */
    channel_dev.flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    if (skip_flag) return;

    if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel) {
        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

        if (!is_exit) {
            /* allocate syn_ops_counter for each block */
            int num_block = p->gridDimX * p->gridDimY * p->gridDimZ;
            CUDA_SAFECALL(hipMalloc(&syn_ops_counter, num_block * sizeof(int)));
            CUDA_SAFECALL(hipMemset(syn_ops_counter, 0, num_block * sizeof(int)));

            /* init race checker */
            race_checker = Checker();
            
            instrument_function_if_needed(ctx, p->f);

            nvbit_enable_instrumented(ctx, p->f, true);

            recv_thread_receiving = true;

            if (DEBUG) {
                int nregs;
                CUDA_SAFECALL(
                    hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

                int shmem_static_nbytes;
                CUDA_SAFECALL(
                    hipFuncGetAttribute(&shmem_static_nbytes,
                                    HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

                printf(
                    "Kernel %s - grid size %d,%d,%d - block size %d,%d,%d - nregs "
                    "%d - shmem %d - cuda stream id %ld\n",
                    nvbit_get_func_name(ctx, p->f), p->gridDimX, p->gridDimY,
                    p->gridDimZ, p->blockDimX, p->blockDimY, p->blockDimZ, nregs,
                    shmem_static_nbytes + p->sharedMemBytes, (uint64_t)p->hStream);
            }
        } else {
            /* make sure current kernel is completed */
            hipDeviceSynchronize();
            assert(hipGetLastError() == hipSuccess);

            /* free allocated cuda memory */
             CUDA_SAFECALL(hipFree(syn_ops_counter));

            /* make sure we prevent re-entry on the nvbit_callback when issuing
             * the flush_channel kernel */
            skip_flag = true;

            /* issue flush of channel so we are sure all the memory accesses
             * have been pushed */
            flush_channel<<<1, 1>>>();
            hipDeviceSynchronize();
            assert(hipGetLastError() == hipSuccess);

            /* unset the skip flag */
            skip_flag = false;

            /* wait here until the receiving thread has not finished with the
             * current kernel */
            while (recv_thread_receiving) {
                pthread_yield();
            }

            printf("\n#kernelends#\n");
        }
    }
}

void *recv_thread_fun(void *) {
    char *recv_buffer = (char *)malloc(CHANNEL_SIZE);

    while (recv_thread_started) {
        uint32_t num_recv_bytes = 0;
        if (recv_thread_receiving &&
            (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
                0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {
                mem_access_t *ma =
                    (mem_access_t *)&recv_buffer[num_processed_bytes];

                /* when we get this block_id it means the kernel has completed
                 */
                if (ma->block_id == -1) {
                    recv_thread_receiving = false;
                    break;
                }
                
                race_checker.read(ma);
                
                num_processed_bytes += sizeof(mem_access_t);
            }
        }
    }
    free(recv_buffer);
    return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
    recv_thread_started = true;
    channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
    pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    if (recv_thread_started) {
        recv_thread_started = false;
        pthread_join(recv_thread, NULL);
    }
}
